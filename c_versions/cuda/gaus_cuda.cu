#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define REAL double

__global__ void gaussianEliminationKernel(REAL* A, REAL* b, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int pivot = blockIdx.y;

    if (row > pivot && row < n) {
        REAL coeff = A[row * n + pivot] / A[pivot * n + pivot];
        for (int col = pivot; col < n; col++) {
            A[row * n + col] -= A[pivot * n + col] * coeff;
        }
        b[row] -= b[pivot] * coeff;
    }
}

__global__ void initializeSystemKernel(REAL* A, REAL* b, int n, int triangular_mode) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n * n) {
        int row = idx / n;
        int col = idx % n;

        if (!triangular_mode || col >= row) {
            if (row == col) {
                A[idx] = n / 10.0;
            } else {
                A[idx] = (row + col) % 5 + 1;
            }
        } else {
            A[idx] = 0.0;
        }
    }

    if (idx < n) {
        b[idx] = n;
    }
}

__global__ void backSubstitutionKernel(REAL* A, REAL* b, REAL* x, int n) {
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < n) {
        REAL sum = b[row];
        for (int col = n - 1; col > row; col--) {
            sum -= A[row * n + col] * x[col];
        }
        x[row] = sum / A[row * n + row];
    }
}

void gaussian_elimination(REAL* d_A, REAL* d_b, int n) {
    dim3 blocks((n + 15) / 16, n);
    dim3 threads(16, 1);
    gaussianEliminationKernel<<<blocks, threads>>>(d_A, d_b, n);
    hipDeviceSynchronize();
}

int main() {
    int n = 10;
    size_t bytes = n * n * sizeof(REAL);
    REAL* h_A = (REAL*)malloc(bytes);
    REAL* h_b = (REAL*)malloc(n * sizeof(REAL));

    REAL* d_A; hipMalloc(&d_A, bytes);
    REAL* d_b; hipMalloc(&d_b, n * sizeof(REAL));

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * sizeof(REAL), hipMemcpyHostToDevice);

    gaussian_elimination(d_A, d_b, n);

    hipMemcpy(h_A, d_A, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_b, d_b, n * sizeof(REAL), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_b);
    free(h_A);
    free(h_b);

    return 0;
}
